#include "hip/hip_runtime.h"
#include "rsbench.cuh"

////////////////////////////////////////////////////////////////////////////////////
// BASELINE FUNCTIONS
////////////////////////////////////////////////////////////////////////////////////
// All "baseline" code is at the top of this file. The baseline code is a simple
// implementation of the algorithm, with only minor GPU optimizations in place.
// Following these functions are a number of optimized variants,
// which each deploy a different combination of optimizations strategies. By
// default, RSBench will only run the baseline implementation. Optimized variants
// must be specifically selected using the "-k <optimized variant ID>" command
// line argument.
////////////////////////////////////////////////////////////////////////////////////

void run_event_based_simulation(Input input, SimulationData GSD, unsigned long * vhash_result )
{
	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	printf("Running baseline event-based simulation on device...\n");

	int nthreads = 256;
	int nblocks = ceil( (double) input.lookups / (double) nthreads);

	xs_lookup_kernel_baseline<<<nblocks, nthreads>>>( input, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + input.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	*vhash_result = verification_scalar;
}

// In this kernel, we perform a single lookup with each thread. Threads within a warp
// do not really have any relation to each other, and divergence due to high nuclide count fuel
// material lookups are costly. This kernel constitutes baseline performance.
__global__ void xs_lookup_kernel_baseline(Input in, SimulationData GSD )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	const int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= in.lookups )
		return;

	// Set the initial seed value
	uint64_t seed = STARTING_SEED;	

	// Forward seed to lookup index (we need 2 samples per lookup)
	seed = fast_forward_LCG(seed, 2*i);

	// Randomly pick an energy and material for the particle
	double E = LCG_random_double(&seed);
	int mat  = pick_mat(&seed);

	double macro_xs[4] = {0};

	calculate_macro_xs( macro_xs, mat, E, in, GSD.num_nucs, GSD.mats, GSD.max_num_nucs, GSD.concs, GSD.n_windows, GSD.pseudo_K0RS, GSD.windows, GSD.poles, GSD.max_num_windows, GSD.max_num_poles );

	// For verification, and to prevent the compiler from optimizing
	// all work out, we interrogate the returned macro_xs_vector array
	// to find its maximum value index, then increment the verification
	// value by that index. In this implementation, we write to a global
	// verification array that will get reduced after this kernel comples.
	double max = -DBL_MAX;
	int max_idx = 0;
	for(int x = 0; x < 4; x++ )
	{
		if( macro_xs[x] > max )
		{
			max = macro_xs[x];
			max_idx = x;
		}
	}
	GSD.verification[i] = max_idx+1;
}

__device__ void calculate_macro_xs( double * macro_xs, int mat, double E, Input input, int * num_nucs, int * mats, int max_num_nucs, double * concs, int * n_windows, double * pseudo_K0Rs, Window * windows, Pole * poles, int max_num_windows, int max_num_poles ) 
{
	// zero out macro vector
	for( int i = 0; i < 4; i++ )
		macro_xs[i] = 0;

	// for nuclide in mat
	for( int i = 0; i < num_nucs[mat]; i++ )
	{
		double micro_xs[4];
		int nuc = mats[mat * max_num_nucs + i];

		if( input.doppler == 1 )
			calculate_micro_xs_doppler( micro_xs, nuc, E, input, n_windows, pseudo_K0Rs, windows, poles, max_num_windows, max_num_poles);
		else
			calculate_micro_xs( micro_xs, nuc, E, input, n_windows, pseudo_K0Rs, windows, poles, max_num_windows, max_num_poles);

		for( int j = 0; j < 4; j++ )
		{
			macro_xs[j] += micro_xs[j] * concs[mat * max_num_nucs + i];
		}
		// Debug
		/*
		printf("E = %.2lf, mat = %d, macro_xs[0] = %.2lf, macro_xs[1] = %.2lf, macro_xs[2] = %.2lf, macro_xs[3] = %.2lf\n",
		E, mat, macro_xs[0], macro_xs[1], macro_xs[2], macro_xs[3] );
		*/
	}

	// Debug
	/*
	printf("E = %.2lf, mat = %d, macro_xs[0] = %.2lf, macro_xs[1] = %.2lf, macro_xs[2] = %.2lf, macro_xs[3] = %.2lf\n",
	E, mat, macro_xs[0], macro_xs[1], macro_xs[2], macro_xs[3] );
	*/
}

// No Temperature dependence (i.e., 0K evaluation)
__device__ void calculate_micro_xs( double * micro_xs, int nuc, double E, Input input, int * n_windows, double * pseudo_K0RS, Window * windows, Pole * poles, int max_num_windows, int max_num_poles)
{
	// MicroScopic XS's to Calculate
	double sigT;
	double sigA;
	double sigF;
	double sigE;

	// Calculate Window Index
	double spacing = 1.0 / n_windows[nuc];
	int window = (int) ( E / spacing );
	if( window == n_windows[nuc] )
		window--;

	// Calculate sigTfactors
	RSComplex sigTfactors[4]; // Of length input.numL, which is always 4
	calculate_sig_T(nuc, E, input, pseudo_K0RS, sigTfactors );

	// Calculate contributions from window "background" (i.e., poles outside window (pre-calculated)
	Window w = windows[nuc * max_num_windows + window];
	sigT = E * w.T;
	sigA = E * w.A;
	sigF = E * w.F;

	// Loop over Poles within window, add contributions
	for( int i = w.start; i < w.end; i++ )
	{
		RSComplex PSIIKI;
		RSComplex CDUM;
		Pole pole = poles[nuc * max_num_poles + i];
		RSComplex t1 = {0, 1};
		RSComplex t2 = {sqrt(E), 0 };
		PSIIKI = c_div( t1 , c_sub(pole.MP_EA,t2) );
		RSComplex E_c = {E, 0};
		CDUM = c_div(PSIIKI, E_c);
		sigT += (c_mul(pole.MP_RT, c_mul(CDUM, sigTfactors[pole.l_value])) ).r;
		sigA += (c_mul( pole.MP_RA, CDUM)).r;
		sigF += (c_mul(pole.MP_RF, CDUM)).r;
	}

	sigE = sigT - sigA;

	micro_xs[0] = sigT;
	micro_xs[1] = sigA;
	micro_xs[2] = sigF;
	micro_xs[3] = sigE;
}

// Temperature Dependent Variation of Kernel
// (This involves using the Complex Faddeeva function to
// Doppler broaden the poles within the window)
__device__ void calculate_micro_xs_doppler( double * micro_xs, int nuc, double E, Input input, int * n_windows, double * pseudo_K0RS, Window * windows, Pole * poles, int max_num_windows, int max_num_poles )
{
	// MicroScopic XS's to Calculate
	double sigT;
	double sigA;
	double sigF;
	double sigE;

	// Calculate Window Index
	double spacing = 1.0 / n_windows[nuc];
	int window = (int) ( E / spacing );
	if( window == n_windows[nuc] )
		window--;

	// Calculate sigTfactors
	RSComplex sigTfactors[4]; // Of length input.numL, which is always 4
	calculate_sig_T(nuc, E, input, pseudo_K0RS, sigTfactors );

	// Calculate contributions from window "background" (i.e., poles outside window (pre-calculated)
	Window w = windows[nuc * max_num_windows + window];
	sigT = E * w.T;
	sigA = E * w.A;
	sigF = E * w.F;

	double dopp = 0.5;

	// Loop over Poles within window, add contributions
	for( int i = w.start; i < w.end; i++ )
	{
		Pole pole = poles[nuc * max_num_poles + i];

		// Prep Z
		RSComplex E_c = {E, 0};
		RSComplex dopp_c = {dopp, 0};
		RSComplex Z = c_mul(c_sub(E_c, pole.MP_EA), dopp_c);

		// Evaluate Fadeeva Function
		RSComplex faddeeva = fast_nuclear_W( Z );

		// Update W
		sigT += (c_mul( pole.MP_RT, c_mul(faddeeva, sigTfactors[pole.l_value]) )).r;
		sigA += (c_mul( pole.MP_RA , faddeeva)).r;
		sigF += (c_mul( pole.MP_RF , faddeeva)).r;
	}

	sigE = sigT - sigA;

	micro_xs[0] = sigT;
	micro_xs[1] = sigA;
	micro_xs[2] = sigF;
	micro_xs[3] = sigE;
}

// picks a material based on a probabilistic distribution
__device__ int pick_mat( uint64_t * seed )
{
	// I have a nice spreadsheet supporting these numbers. They are
	// the fractions (by volume) of material in the core. Not a 
	// *perfect* approximation of where XS lookups are going to occur,
	// but this will do a good job of biasing the system nonetheless.

	double dist[12];
	dist[0]  = 0.140;	// fuel
	dist[1]  = 0.052;	// cladding
	dist[2]  = 0.275;	// cold, borated water
	dist[3]  = 0.134;	// hot, borated water
	dist[4]  = 0.154;	// RPV
	dist[5]  = 0.064;	// Lower, radial reflector
	dist[6]  = 0.066;	// Upper reflector / top plate
	dist[7]  = 0.055;	// bottom plate
	dist[8]  = 0.008;	// bottom nozzle
	dist[9]  = 0.015;	// top nozzle
	dist[10] = 0.025;	// top of fuel assemblies
	dist[11] = 0.013;	// bottom of fuel assemblies

	double roll = LCG_random_double(seed);

	// makes a pick based on the distro
	for( int i = 0; i < 12; i++ )
	{
		double running = 0;
		for( int j = i; j > 0; j-- )
			running += dist[j];
		if( roll < running )
			return i;
	}

	return 0;
}

__device__ void calculate_sig_T( int nuc, double E, Input input, double * pseudo_K0RS, RSComplex * sigTfactors )
{
	double phi;

	for( int i = 0; i < 4; i++ )
	{
		phi = pseudo_K0RS[nuc * input.numL + i] * sqrt(E);

		if( i == 1 )
			phi -= - atan( phi );
		else if( i == 2 )
			phi -= atan( 3.0 * phi / (3.0 - phi*phi));
		else if( i == 3 )
			phi -= atan(phi*(15.0-phi*phi)/(15.0-6.0*phi*phi));

		phi *= 2.0;

		sigTfactors[i].r = cos(phi);
		sigTfactors[i].i = -sin(phi);
	}
}

// This function uses a combination of the Abrarov Approximation
// and the QUICK_W three term asymptotic expansion.
// Only expected to use Abrarov ~0.5% of the time.
__device__ RSComplex fast_nuclear_W( RSComplex Z )
{
	// Abrarov 
	if( c_abs(Z) < 6.0 )
	{
		// Precomputed parts for speeding things up
		// (N = 10, Tm = 12.0)
		RSComplex prefactor = {0, 8.124330e+01};
		double an[10] = {
			2.758402e-01,
			2.245740e-01,
			1.594149e-01,
			9.866577e-02,
			5.324414e-02,
			2.505215e-02,
			1.027747e-02,
			3.676164e-03,
			1.146494e-03,
			3.117570e-04
		};
		double neg_1n[10] = {
			-1.0,
			1.0,
			-1.0,
			1.0,
			-1.0,
			1.0,
			-1.0,
			1.0,
			-1.0,
			1.0
		};

		double denominator_left[10] = {
			9.869604e+00,
			3.947842e+01,
			8.882644e+01,
			1.579137e+02,
			2.467401e+02,
			3.553058e+02,
			4.836106e+02,
			6.316547e+02,
			7.994380e+02,
			9.869604e+02
		};

		RSComplex t1 = {0, 12};
		RSComplex t2 = {12, 0};
		RSComplex i = {0,1};
		RSComplex one = {1, 0};
		RSComplex W = c_div(c_mul(i, ( c_sub(one, fast_cexp(c_mul(t1, Z))) )) , c_mul(t2, Z));
		RSComplex sum = {0,0};
		for( int n = 0; n < 10; n++ )
		{
			RSComplex t3 = {neg_1n[n], 0};
			RSComplex top = c_sub(c_mul(t3, fast_cexp(c_mul(t1, Z))), one);
			RSComplex t4 = {denominator_left[n], 0};
			RSComplex t5 = {144, 0};
			RSComplex bot = c_sub(t4, c_mul(t5,c_mul(Z,Z)));
			RSComplex t6 = {an[n], 0};
			sum = c_add(sum, c_mul(t6, c_div(top,bot)));
		}
		W = c_add(W, c_mul(prefactor, c_mul(Z, sum)));
		return W;
	}
	else
	{
		// QUICK_2 3 Term Asymptotic Expansion (Accurate to O(1e-6)).
		// Pre-computed parameters
		RSComplex a = {0.512424224754768462984202823134979415014943561548661637413182,0};
		RSComplex b = {0.275255128608410950901357962647054304017026259671664935783653, 0};
		RSComplex c = {0.051765358792987823963876628425793170829107067780337219430904, 0};
		RSComplex d = {2.724744871391589049098642037352945695982973740328335064216346, 0};

		RSComplex i = {0,1};
		RSComplex Z2 = c_mul(Z, Z);
		// Three Term Asymptotic Expansion
		RSComplex W = c_mul(c_mul(Z,i), (c_add(c_div(a,(c_sub(Z2, b))) , c_div(c,(c_sub(Z2, d))))));

		return W;
	}
}

__host__ __device__ double LCG_random_double(uint64_t * seed)
{
	const uint64_t m = 9223372036854775808ULL; // 2^63
	const uint64_t a = 2806196910506780709ULL;
	const uint64_t c = 1ULL;
	*seed = (a * (*seed) + c) % m;
	return (double) (*seed) / (double) m;
}	

__host__ __device__ uint64_t LCG_random_int(uint64_t * seed)
{
	const uint64_t m = 9223372036854775808ULL; // 2^63
	const uint64_t a = 2806196910506780709ULL;
	const uint64_t c = 1ULL;
	*seed = (a * (*seed) + c) % m;
	return *seed;
}	

__device__ uint64_t fast_forward_LCG(uint64_t seed, uint64_t n)
{
	const uint64_t m = 9223372036854775808ULL; // 2^63
	uint64_t a = 2806196910506780709ULL;
	uint64_t c = 1ULL;

	n = n % m;

	uint64_t a_new = 1;
	uint64_t c_new = 0;

	while(n > 0) 
	{
		if(n & 1)
		{
			a_new *= a;
			c_new = c_new * a + c;
		}
		c *= (a + 1);
		a *= a;

		n >>= 1;
	}

	return (a_new * seed + c_new) % m;
}

// Complex arithmetic functions

__device__ RSComplex c_add( RSComplex A, RSComplex B)
{
	RSComplex C;
	C.r = A.r + B.r;
	C.i = A.i + B.i;
	return C;
}

__device__ RSComplex c_sub( RSComplex A, RSComplex B)
{
	RSComplex C;
	C.r = A.r - B.r;
	C.i = A.i - B.i;
	return C;
}

__host__ __device__ RSComplex c_mul( RSComplex A, RSComplex B)
{
	double a = A.r;
	double b = A.i;
	double c = B.r;
	double d = B.i;
	RSComplex C;
	C.r = (a*c) - (b*d);
	C.i = (a*d) + (b*c);
	return C;
}

__device__ RSComplex c_div( RSComplex A, RSComplex B)
{
	double a = A.r;
	double b = A.i;
	double c = B.r;
	double d = B.i;
	RSComplex C;
	double denom = c*c + d*d;
	C.r = ( (a*c) + (b*d) ) / denom;
	C.i = ( (b*c) - (a*d) ) / denom;
	return C;
}

__device__ double c_abs( RSComplex A)
{
	return sqrt(A.r*A.r + A.i * A.i);
}


// Fast (but inaccurate) exponential function
// Written By "ACMer":
// https://codingforspeed.com/using-faster-exponential-approximation/
// We use our own to avoid small differences in compiler specific
// exp() intrinsic implementations that make it difficult to verify
// if the code is working correctly or not.
__device__ double fast_exp(double x)
{
  x = 1.0 + x * 0.000244140625;
  x *= x; x *= x; x *= x; x *= x;
  x *= x; x *= x; x *= x; x *= x;
  x *= x; x *= x; x *= x; x *= x;
  return x;
}

// Implementation based on:
// z = x + iy
// cexp(z) = e^x * (cos(y) + i * sin(y))
__device__ RSComplex fast_cexp( RSComplex z )
{
	double x = z.r;
	double y = z.i;

	// For consistency across architectures, we
	// will use our own exponetial implementation
	//double t1 = exp(x);
	double t1 = fast_exp(x);
	double t2 = cos(y);
	double t3 = sin(y);
	RSComplex t4 = {t2, t3};
	RSComplex t5 = {t1, 0};
	RSComplex result = c_mul(t5, (t4));
	return result;
}	

////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
// OPTIMIZED VARIANT FUNCTIONS
////////////////////////////////////////////////////////////////////////////////////
// This section contains a number of optimized variants of some of the above
// functions, which each deploy a different combination of optimizations strategies
// specific to GPU. By default, RSBench will not run any of these variants. They
// must be specifically selected using the "-k <optimized variant ID>" command
// line argument.
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////
// Optimization 6 -- Kernel Splitting + All Material Lookups + Full Sort
//                   + Energy Sort
////////////////////////////////////////////////////////////////////////////////////
// This optimization builds on optimization 4, adding in a second sort by energy.
// It is extremely fast, as now most of the threads within a warp will be hitting
// the same indices in the lookup grids. This greatly reduces thread divergence and
// greatly improves cache efficiency and re-use.
//
// However, it is unlikely that this exact optimization would be possible in a real
// application like OpenMC. One major difference is that particle objects are quite
// large, often having 50+ variable fields, such that sorting them in memory becomes
// rather expensive. Instead, the best possible option would probably be to create
// intermediate indexing (per Hamilton et. al 2019), and run the kernels indirectly.
////////////////////////////////////////////////////////////////////////////////////

__global__ void sampling_kernel(Input in, SimulationData GSD )
{
	// The lookup ID.
	const int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= in.lookups )
		return;

	// Set the initial seed value
	uint64_t seed = STARTING_SEED;	

	// Forward seed to lookup index (we need 2 samples per lookup)
	seed = fast_forward_LCG(seed, 2*i);

	// Randomly pick an energy and material for the particle
	double p_energy = LCG_random_double(&seed);
	int mat         = pick_mat(&seed); 

	// Store sample data in state array
	GSD.p_energy_samples[i] = p_energy;
	GSD.mat_samples[i] = mat;
}

__global__ void xs_lookup_kernel_optimization_1(Input in, SimulationData GSD, int m, int n_lookups, int offset )
{
	// The lookup ID. Used to set the seed, and to store the verification value
	int i = blockIdx.x *blockDim.x + threadIdx.x;

	if( i >= n_lookups )
		return;

	i += offset;

	// Check that our material type matches the kernel material
	int mat = GSD.mat_samples[i];
	if( mat != m )
		return;
	
	double macro_xs[4] = {0};

	calculate_macro_xs( macro_xs, mat, GSD.p_energy_samples[i], in, GSD.num_nucs, GSD.mats, GSD.max_num_nucs, GSD.concs, GSD.n_windows, GSD.pseudo_K0RS, GSD.windows, GSD.poles, GSD.max_num_windows, GSD.max_num_poles );

	// For verification, and to prevent the compiler from optimizing
	// all work out, we interrogate the returned macro_xs_vector array
	// to find its maximum value index, then increment the verification
	// value by that index. In this implementation, we write to a global
	// verification array that will get reduced after this kernel comples.
	double max = -DBL_MAX;
	int max_idx = 0;
	for(int x = 0; x < 4; x++ )
	{
		if( macro_xs[x] > max )
		{
			max = macro_xs[x];
			max_idx = x;
		}
	}
	GSD.verification[i] = max_idx+1;
}

void run_event_based_simulation_optimization_1(Input in, SimulationData GSD, unsigned long * vhash_result)
{
	const char * optimization_name = "Optimization 1 - Material & Energy Sorts + Material-specific Kernels";
	
	printf("Simulation Kernel:\"%s\"\n", optimization_name);
	
	////////////////////////////////////////////////////////////////////////////////
	// Allocate Additional Data Structures Needed by Optimized Kernel
	////////////////////////////////////////////////////////////////////////////////
	printf("Allocating additional device data required by kernel...\n");
	size_t sz;
	size_t total_sz = 0;

	sz = in.lookups * sizeof(double);
	gpuErrchk( hipMalloc((void **) &GSD.p_energy_samples, sz) );
	total_sz += sz;
	GSD.length_p_energy_samples = in.lookups;

	sz = in.lookups * sizeof(int);
	gpuErrchk( hipMalloc((void **) &GSD.mat_samples, sz) );
	total_sz += sz;
	GSD.length_mat_samples = in.lookups;
	
	printf("Allocated an additional %.0lf MB of data on GPU.\n", total_sz/1024.0/1024.0);

	////////////////////////////////////////////////////////////////////////////////
	// Configure & Launch Simulation Kernel
	////////////////////////////////////////////////////////////////////////////////
	printf("Beginning optimized simulation...\n");

	int nthreads = 32;
	int nblocks = ceil( (double) in.lookups / 32.0);
	
	sampling_kernel<<<nblocks, nthreads>>>( in, GSD );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	// Count the number of fuel material lookups that need to be performed (fuel id = 0)
	int n_lookups_per_material[12];
	for( int m = 0; m < 12; m++ )
		n_lookups_per_material[m] = thrust::count(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, m);

	// Sort by material first
	thrust::sort_by_key(thrust::device, GSD.mat_samples, GSD.mat_samples + in.lookups, GSD.p_energy_samples);

	// Now, sort each material by energy
	int offset = 0;
	for( int m = 0; m < 12; m++ )
	{
		thrust::sort_by_key(thrust::device, GSD.p_energy_samples + offset, GSD.p_energy_samples + offset + n_lookups_per_material[m], GSD.mat_samples + offset);
		offset += n_lookups_per_material[m];
	}
	
	// Launch all material kernels individually
	offset = 0;
	for( int m = 0; m < 12; m++ )
	{
		nthreads = 32;
		nblocks = ceil((double) n_lookups_per_material[m] / (double) nthreads);
		xs_lookup_kernel_optimization_1<<<nblocks, nthreads>>>( in, GSD, m, n_lookups_per_material[m], offset );
		offset += n_lookups_per_material[m];
	}
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	////////////////////////////////////////////////////////////////////////////////
	// Reduce Verification Results
	////////////////////////////////////////////////////////////////////////////////
	printf("Reducing verification results...\n");

	unsigned long verification_scalar = thrust::reduce(thrust::device, GSD.verification, GSD.verification + in.lookups, 0);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	*vhash_result = verification_scalar;
}
