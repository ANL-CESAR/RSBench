#include "hip/hip_runtime.h"
#include "rsbench.h"
#include "My_Stats.h"

__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val +
					__longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__device__  void calc_sig_T ( int i, double phi, hipDoubleComplex* rslt) {
	if ( i == 1 )
		phi -= atan ( phi );
	else if( i == 2 )
		phi -= atan ( 3.0 * phi / (3.0 - phi*phi));
	else if( i == 3 )
		phi -= atan (phi*(15.0-phi*phi)/(15.0-6.0*phi*phi));

	phi += phi;

	rslt -> x = cos(phi); 
	rslt -> y = sin(phi);
}

__global__ void calc_sig_T_sim_kernel ( double E, int num_iter, 
		const double* data, hipDoubleComplex* gTfactors) {
	calc_sig_T (threadIdx.x, data[threadIdx.x] * sqrt(E), &gTfactors[threadIdx.x]);	
}

/*
__global__ void (CalcDataPtrs_d* data) {
	double micro_xs[4];
	int nuc = data->materials.mats_2d[mat* data->materials.pitch + i ];

	// MicroScopic XS's to Calculate
	double sigT, sigA, sigF, sigE;

	// Calculate Window Index
	double spacing = 1.0 / data->n_windows[nuc];
	int window = (int) ( E / spacing );
	if( window == data->n_windows[nuc] )
		window--;

	// Calculate sigTfactors
	calculate_sig_T_sim ( E, input.numL, data->pseudo_K0RS_2d[nuc], sigTfactors );
	// Calculate contributions from window "background" (i.e., poles outside window (pre-calculated)
	Window w = data->windows[nuc][window];
	sigT = E * w.T;
	sigA = E * w.A;
	sigF = E * w.F;
	// Loop over Poles within window, add contributions
	hipDoubleComplex const1 = make_hipDoubleComplex(0, 1/E), const2 = make_hipDoubleComplex( sqrt(E), 0);
	for( int i = w.start; i < w.end; i++ )	{
		Pole pole = data->poles_2d[nuc+i];
		hipDoubleComplex CDUM = hipCdiv( const1, hipCsub( pole.MP_EA, const2 ) );
		sigT += hipCreal( hipCmul( pole.MP_RT, hipCmul( CDUM, sigTfactors[pole.l_value] ) ) );
		sigA += hipCreal( hipCmul( pole.MP_RA, CDUM) );
		sigF += hipCreal( hipCmul( pole.MP_RF, CDUM) );
	}

	sigE = sigT - sigA;

	micro_xs[0] = sigT;	micro_xs[1] = sigA;	micro_xs[2] = sigF;	micro_xs[3] = sigE;

	for( int j = 0; j < 4; j++ ){
		macro_xs[j] += micro_xs[j] * data->materials.concs_2d[mat+i];
	}
}*/
// use data_d
void calculate_macro_xs_d ( double * macro_xs, int mat, double E, Input input, CalcDataPtrs_d* data, hipDoubleComplex * sigTfactors ) {
	// zero out macro vector
	for( int i = 0; i < 4; i++ )
		macro_xs[i] = 0;

	// for nuclide in mat
	for( int i = 0; i < data->materials.num_nucs[mat]; i++ ){
	}
}

__global__ void calc_sig_kernel ( hipDoubleComplex const1, hipDoubleComplex const2, Pole* poles, 
		int base, double* sigT, double* sigA, double* sigF, hipDoubleComplex* sigTfactors) {
	Pole pole = poles[blockIdx.x + base ];
	hipDoubleComplex CDUM = hipCdiv( const1, hipCsub( pole.MP_EA, const2 ) );
	sigT[blockIdx.x + base] = hipCreal( hipCmul( pole.MP_RT, hipCmul( CDUM, sigTfactors[pole.l_value] ) ) );
	sigA[blockIdx.x + base] = hipCreal( hipCmul( pole.MP_RA, CDUM) );
	sigF[blockIdx.x + base] = hipCreal( hipCmul( pole.MP_RF, CDUM) );
}

__global__ void sum_sigs ( double* sigTs, double* sigAs, double* sigFs, double* sigT, double* sigA, double* sigF, 
		int tpb, int len ) {
	*sigA = *sigF = *sigT = 0;
	int i, j;
	for ( i = 0; i < len; i += tpb ) {
		if ( ( j = i + threadIdx.x ) < len ) {
			atomicAdd(sigA, sigAs[j]);
			atomicAdd(sigT, sigTs[j]);
			atomicAdd(sigF, sigFs[j]);
		}
	}
}

//	CUDA adaptation of 
void calc_sig_driver ( double * micro_xs, int nuc, double E, Input input, CalcDataPtrs data, hipDoubleComplex * sigTfactors ) {
	// MicroScopic XS's to Calculate
	double sigT, sigA, sigF, sigE;

	// Calculate Window Index
	double spacing = 1.0 / data.n_windows[nuc];
	int window = (int) ( E / spacing );
	if( window == data.n_windows[nuc] )
		window--;

	// Calculate sigTfactors
	calculate_sig_T_sim ( E, input.numL, data.pseudo_K0RS[nuc], sigTfactors );

	// Calculate contributions from window "background" (i.e., poles outside window (pre-calculated)
	Window w = data.windows[nuc][window];
	sigT = E * w.T;	sigA = E * w.A;	sigF = E * w.F;
	// Loop over Poles within window, add contributions
	hipDoubleComplex const1 = make_hipDoubleComplex(0, 1/E), const2 = make_hipDoubleComplex( sqrt(E), 0);
	int num = w.end - w.start + 1;
	double* sigTs, *sigAs, *sigFs, *sigT_d, *sigA_d, *sigF_d; 
	Pole* poles_d; 
	hipDoubleComplex* sigTfactors_d;
	/* allocate memory on device */
	assert (hipMalloc((void **) &sigTs, num*sizeof(double)) == hipSuccess);
	assert (hipMalloc((void **) &sigAs, num*sizeof(double)) == hipSuccess);
	assert (hipMalloc((void **) &sigFs, num*sizeof(double)) == hipSuccess);
	assert (hipMalloc((void **) &sigT_d, sizeof(double)) == hipSuccess);
	assert (hipMalloc((void **) &sigA_d, sizeof(double)) == hipSuccess);
	assert (hipMalloc((void **) &sigF_d, sizeof(double)) == hipSuccess);
	assert (hipMalloc((void **) &poles_d, num*sizeof(Pole)) == hipSuccess);
	assert (hipMalloc((void **) &sigTfactors_d, num*sizeof(hipDoubleComplex)) == hipSuccess);
	assert(hipMemcpy( sigTfactors_d, sigTfactors, input.numL*sizeof(hipDoubleComplex),hipMemcpyHostToDevice) == hipSuccess);
	assert(hipMemcpy( poles_d, data.poles[nuc], sizeof(Pole),hipMemcpyHostToDevice) == hipSuccess);
	calc_sig_kernel<<<num, 1>>> ( const1, const2, poles_d, 
			w.start, sigTs, sigAs, sigFs, sigTfactors_d );
	sum_sigs<<<1, 512>>> ( sigTs, sigAs, sigFs, sigT_d, sigA_d, sigF_d, 512, num );
	assert(hipMemcpy( &sigT, sigT_d, sizeof(double),hipMemcpyDeviceToHost) == hipSuccess);
	assert(hipMemcpy( &sigA, sigA_d, sizeof(double),hipMemcpyDeviceToHost) == hipSuccess);
	assert(hipMemcpy( &sigF, sigF_d, sizeof(double),hipMemcpyDeviceToHost) == hipSuccess);

	hipFree( sigTs );  hipFree( sigAs );  hipFree( sigFs );  hipFree( poles_d); hipFree( sigTfactors_d);
	hipFree( sigT_d );  hipFree( sigA_d );  hipFree( sigF_d );  

	sigE = sigT - sigA;

	micro_xs[0] = sigT;	micro_xs[1] = sigA;	micro_xs[2] = sigF;	micro_xs[3] = sigE;
}

void calculate_micro_xs_driver( double * micro_xs, int nuc, double E, Input input, CalcDataPtrs data, hipDoubleComplex * sigTfactors)
{
	// MicroScopic XS's to Calculate
	double sigT, sigA, sigF, sigE;
	double* data_d;
	hipDoubleComplex* cudcomp_d;

	// Calculate Window Index
	double spacing = 1.0 / data.n_windows[nuc];
	int window = (int) ( E / spacing );
	if( window == data.n_windows[nuc] )
		window--;

	/* allocate memory on device */
	assert (hipMalloc((void **) &data_d, input.numL*sizeof(double)) == hipSuccess);
	assert (hipMalloc((void **) &cudcomp_d, input.numL*sizeof(hipDoubleComplex)) == hipSuccess);
	/* copy host data to device pointers */
	assert(hipMemcpy(data_d, data.pseudo_K0RS[nuc], input.numL*sizeof(double),hipMemcpyHostToDevice) == hipSuccess);
	// Calculate sigTfactors
	calc_sig_T_sim_kernel<<<1, input.numL>>> ( E, input.numL, data_d, cudcomp_d);
	assert(hipMemcpy( sigTfactors, cudcomp_d, input.numL*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost) == hipSuccess);
	hipFree( cudcomp_d );  
	hipFree( data_d );  
	// Calculate contributions from window "background" (i.e., poles outside window (pre-calculated)
	Window w = data.windows[nuc][window];
	sigT = E * w.T;
	sigA = E * w.A;
	sigF = E * w.F;
	// Loop over Poles within window, add contributions
	hipDoubleComplex const1 = make_hipDoubleComplex(0, 1/E), const2 = make_hipDoubleComplex( sqrt(E), 0);
	for( int i = w.start; i < w.end; i++ ) {
		Pole pole = data.poles[nuc][i];
		hipDoubleComplex CDUM = hipCdiv( const1, hipCsub( pole.MP_EA, const2 ) );
		sigT += hipCreal( hipCmul( pole.MP_RT, hipCmul( CDUM, sigTfactors[pole.l_value] ) ) );
		sigA += hipCreal( hipCmul( pole.MP_RA, CDUM) );
		sigF += hipCreal( hipCmul( pole.MP_RF, CDUM) );
	}

	sigE = sigT - sigA;

	micro_xs[0] = sigT;
	micro_xs[1] = sigA;
	micro_xs[2] = sigF;
	micro_xs[3] = sigE;
}

int dotp_driver(int NTPB){
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
				prop.memoryBusWidth);
		printf("  Maximum number of threads per block: %d\n",
				prop.maxThreadsPerBlock);
		printf("  Maximum size of each dimension of a block: %d %d %d\n",
				prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}
	return 0;
}
