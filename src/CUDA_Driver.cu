#include "hip/hip_runtime.h"
#include "rsbench.h"
#include "My_Stats.h"

const unsigned int NUM_ITER = 100;
/* vector lengths */
unsigned int sizes[] = {16384, 65536, 262144, 1048576, 4194304, 16777216, 33554432, 50331648, 67108864, 268435456};

__device__  void calc_sig_T ( int i, double phi, hipDoubleComplex* rslt) {
	if ( i == 1 )
		phi -= atan ( phi );
	else if( i == 2 )
		phi -= atan ( 3.0 * phi / (3.0 - phi*phi));
	else if( i == 3 )
		phi -= atan (phi*(15.0-phi*phi)/(15.0-6.0*phi*phi));

	phi += phi;

	rslt -> x = cos(phi); 
	rslt -> y = sin(phi);
}

__global__ void calc_sig_T_sim_kernel ( double E, int num_iter, const double* data, hipDoubleComplex* gTfactors) {
	calc_sig_T ( threadIdx.x, data[threadIdx.x] * sqrt(E), &gTfactors[threadIdx.x]);	
}

__global__ void dotp(int *a, int *b, int *c, int n){
	int i;
	int iglob = threadIdx.x + blockIdx.x*blockDim.x; 
	int iloc  = threadIdx.x                        ;
	/* each block's pairwise products are stored in this temporary array */
	//  __shared__ int block_cache[NTPB]; 
	//note that our block_cache array can be sized statically with a constant
	//or dynamically with the size in bytes passed in as the third argument
	//to the kernel launch specificer
	extern __shared__ int block_cache[]; 

	if (iglob < n)
		block_cache[iloc] = a[iglob]*b[iglob];
	else
		block_cache[iloc] = 0;

	__syncthreads();

	if (iloc == 0){
		int sum = 0;
		for (i=0;i< blockDim.x;++i)
			sum += block_cache[i];
		atomicAdd(c,sum);  /* now write safely to global memory */
	}
}

int dotp_driver(int NTPB){
	int *a,   *b,   *c;       /* host pointers */
	int *a_d, *b_d, *c_d;     /* device pointers */
	int i, j, k;			/* vector length */
	/* Number of Threads Per Block */
	assert( NTPB > 0 );
	hipEvent_t start, stop;  /* timers */
	float time;

	double trials [NUM_ITER];
	double* stats=NULL;

	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
				prop.memoryBusWidth);
		printf("  Maximum number of threads per block: %d\n",
				prop.maxThreadsPerBlock);
		printf("  Maximum size of each dimension of a block: %d %d %d\n",
				prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}

	hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("Number of trials: %u\t Num of threads per block: %i\n", NUM_ITER, NTPB);
	printf("Vector Size,DotProd,Min,25%%,Median,75%%,Max,Mean,Variance,Stdev\n");
	for ( j = 0; j < sizeof (sizes)/sizeof(unsigned int); j ++ ) {
		/* allocate host memory */
		assert (hipHostMalloc((void **) &a, sizes[j]*sizeof(int)) == hipSuccess);
		assert (hipHostMalloc((void **) &b, sizes[j]*sizeof(int)) == hipSuccess);
		assert (hipHostMalloc((void **) &c, 1*sizeof(int)) == hipSuccess);

		/* initialize vectors to some simple values and initialize
		   scalar result of dot product to zero */
		for (i=0;i<sizes[j];++i){
			a[i] =  1;
			b[i] = -2;
		}

		/* allocate memory on device */
		assert (hipMalloc((void **) &a_d, sizes[j]*sizeof(int)) == hipSuccess);
		assert (hipMalloc((void **) &b_d, sizes[j]*sizeof(int)) == hipSuccess);
		assert (hipMalloc((void **) &c_d, 1*sizeof(int)) == hipSuccess);

		/* copy host data to device pointers */
		assert(hipMemcpy(a_d,a,sizes[j]*sizeof(int),hipMemcpyHostToDevice) == hipSuccess);
		assert(hipMemcpy(b_d,b,sizes[j]*sizeof(int),hipMemcpyHostToDevice) == hipSuccess);

		for (k = 0; k < NUM_ITER; k++ ){
			*c = 0;
			assert(hipMemcpy(c_d,c,1*sizeof(int),hipMemcpyHostToDevice) == hipSuccess);

			/* one way to set kernel launch values is to fix Number of Threads Per Block (NTPB)
			   as a constant and then calculate # of blocks based on input problem size n. Below
			   is a simple formula that carries out this calculation. THere are other ways to do
			   this that are just as good */

			/* launch and time kernel code */
			hipEventRecord( start, 0 );  

			dotp<<<(sizes[j])/NTPB,NTPB,NTPB*sizeof(int)>>>(a_d,b_d,c_d,sizes[j]);

			hipEventRecord( stop, 0 );
			hipEventSynchronize( stop );
			hipEventElapsedTime( &time, start, stop );
			trials[k] = time;
			assert(hipMemcpy(c,c_d,1*sizeof(int),hipMemcpyDeviceToHost) == hipSuccess);
		}
		my_stats ( trials, &NUM_ITER, &stats);
		printf ("%u,%i,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", sizes[j], *c, stats[0], stats[1], 
				stats[2], stats[3], stats[4], stats[5], stats[6], stats[7]);
		//		printf("size of vectors: %u\tvalue on device:%d\ttime elapsed: %f(ms)\n", sizes[j], *c, time);

		hipFree(a_d);  hipFree(b_d);  hipFree(c_d);
		hipHostFree(a);  hipHostFree(b);  hipHostFree(c);
	}
	hipEventDestroy( start );
	hipEventDestroy( stop );
	return 0;
}
